#include "hip/hip_runtime.h"
#include "curas.ch"

__global__ void k_fill(float* data, int nx, int ny, int nz) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < nx*ny*nz; 
         i += blockDim.x * gridDim.x) 
      {

          data[i] = sinf(i/1024.);
      
	}
}

void curas_fill(float* data_h, int nx, int ny, int nz) {

	hipError_t e = hipSuccess;

	float* data_d;
	e = hipMalloc((void**) &data_d, nx*ny*nz*sizeof(float));
	curas_checkerr(e, "allocate device memory");


    // Launch the Vector Add CUDA Kernel
	int threadsPerBlock = 256;
	int stridePerBlock = 256;

	/*int numSMs;*/
	/*hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);*/

	int blocksPerGrid = (nx*ny*nz + threadsPerBlock - 1) / (threadsPerBlock * stridePerBlock);
	/*int blocksPerGrid = 256*numSMs;*/
	/*int threadsPerBlock = (nx*ny*nz)/blocksPerGrid;*/



    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    k_fill<<<blocksPerGrid, threadsPerBlock>>>(data_d, nx, ny, nz);
    e = hipGetLastError();
	curas_checkerr(e, "kernel call");

	e = hipMemcpy(data_h, data_d, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);
	curas_checkerr(e, "copy to host");

	e = hipFree(data_d);
	curas_checkerr(e, "free device memory");


	return;
}
  
void curas_checkerr(hipError_t err, char* msg) {
	if (err != hipSuccess) {
		printf("CUDA Error: %s at %s.\n", hipGetErrorString(err), msg);
		exit(0);
	}
}
